#include "hip/hip_runtime.h"
#include "Kernel.cuh"

#define GLM_FORCE_PURE
#include <>
#include <glm/glm.hpp>

using namespace glm;

#ifndef __HIPCC__
int atomicAdd(int *i, int c){};
#endif

__device__ bool plane_intersect(const Plane &collider, const vec3 &position, float radius, vec3 &penetrationNormal,
								vec3 &penetrationPos, float &penetrationLength)
{
	vec3 projection = collider.position - position;

	penetrationNormal = cross(collider.right, collider.forward);
	penetrationLength = abs(dot(projection, penetrationNormal)) - radius / 2.0f;
	penetrationPos = collider.position - projection;

	return penetrationLength < 0.0f && abs(dot(projection, collider.right)) < collider.size.x &&
		   abs(dot(projection, collider.forward)) < collider.size.y;
}

__device__ vec3 dampVelocity(const Plane &collider, const vec3 &velocity, const vec3 &penetrationNormal, float drag)
{
	const vec3 newV = dot(velocity, penetrationNormal) * penetrationNormal * BOUND_DAMPING +
					  dot(velocity, collider.right) * collider.right * drag +
					  dot(velocity, collider.forward) * collider.forward * drag;
	const vec3 forward = vec3(0.0f, 0.0f, 1.0f);
	const vec3 right = vec3(1.0f, 0.0f, 0.0f);
	const vec3 up = vec3(0.0f, 1.0f, 0.0);

	return dot(newV, forward) * forward + dot(newV, right) * right + dot(newV, up) * up;
}

__device__ inline int getGridPos(int x, int y, int z) { return x + gridDimX * (y + gridDimZ * z); }
__device__ inline int getGridIdx(int gridPos, int i) { return gridPos + i * bucketCapacity; }

__device__ i32vec3 getParticleGridPosition(const vec3 &position, const vec3 &worldMin, const vec3 &worldMax)
{
	const vec3 normalized_position = (position - worldMin) / (worldMax - worldMin);

	// Find buckets to place particles in
	// Could be written vectorized but explicit is easier to understand
	int bucketX = min(max(0, int(normalized_position.x * gridDimX)), gridDimX - 1);
	int bucketY = min(max(0, int(normalized_position.y * gridDimY)), gridDimY - 1);
	int bucketZ = min(max(0, int(normalized_position.z * gridDimZ)), gridDimZ - 1);
	return {bucketX, bucketY, bucketZ};
}

__global__ void clearGrid(int *gridCounter, int *gridIndices)
{
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned idy = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned idz = blockIdx.z * blockDim.z + threadIdx.z;
	if (idx >= gridDimX || idy >= gridDimY || idz >= gridDimZ)
		return;

	const int gridPos = getGridPos(idx, idy, idz);
	gridCounter[gridPos] = 0;
}

__global__ void buildGrid(Particle *particles, int particleCount, int *gridCounter, int *gridIndices, vec3 worldMin,
						  vec3 worldMax)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= particleCount)
		return;

	const auto &particle = particles[idx];
	const auto pGridPos = getParticleGridPosition(particle.position, worldMin, worldMax);

	const int gridPos = getGridPos(pGridPos.x, pGridPos.y, pGridPos.z);
	const int i = atomicAdd(&gridCounter[gridPos], 1);

	if (i < bucketCapacity)
		gridIndices[getGridIdx(gridPos, i)] = idx;
}

__global__ void computeDensityPressure(Particle *particles, SimulationParams params, Plane *planes, int particleCount,
									   int planeCount, float timestep, int *gridCounter, int *gridIndices,
									   vec3 worldMin, vec3 worldMax)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= particleCount)
		return;

	auto &pi = particles[idx];
	const i32vec3 piGridPos = getParticleGridPosition(pi.position, worldMin, worldMax);
	const int gridIdx = getGridPos(piGridPos.x, piGridPos.y, piGridPos.z);
	const int counter = gridCounter[gridIdx];

	for (int i = 0; i < counter && i < bucketCapacity; i++)
	{
		const auto &pj = particles[gridIndices[getGridIdx(gridIdx, i)]];
		const vec3 rij = pj.position - pi.position;
		const float r2 = dot(rij, rij);
		if (r2 < params.smoothingRadiusPow2)
		{
			const float temp = params.smoothingRadiusPow2 - r2;
			const float smoothingRadiusMinR2pow3 = temp * temp * temp;
			pi.density += params.particleMass * (315.0f / (64.0f * glm::pi<float>() * params.smoothingRadiusPow9)) *
						  smoothingRadiusMinR2pow3;
		}
	}

	pi.pressure = GAS_CONST * (pi.density * params.restDensity);
}
__global__ void computeForces(Particle *particles, SimulationParams params, Plane *planes, int particleCount,
							  int planeCount, float timestep, int *gridCounter, int *gridIndices, vec3 worldMin,
							  vec3 worldMax)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= particleCount)
		return;

	auto &pi = particles[idx];
	const i32vec3 piGridPos = getParticleGridPosition(pi.position, worldMin, worldMax);

	vec3 forcePressure = vec3(0.0f);
	vec3 forceViscosity = vec3(0.0f);

	const i32vec3 begin = max(i32vec3{0, 0, 0}, piGridPos - 1);
	const i32vec3 end = min(i32vec3{gridDimX - 1, gridDimY - 1, gridDimZ - 1}, piGridPos + 1);

	for (int i = begin.x; i <= end.x; i++)
	{
		for (int j = begin.y; j <= end.y; j++)
		{
			for (int k = begin.z; k <= end.z; k++)
			{
				const int gridj = getGridPos(i, j, k);
				for (int pjIdx = 0; pjIdx < gridCounter[gridj] && pjIdx < bucketCapacity; pjIdx++)
				{
					const auto &pj = particles[gridIndices[getGridIdx(gridj, pjIdx)]];
					if (&pi == &pj)
						continue;

					const vec3 rij = pj.position - pi.position;
					const float r2 = dot(rij, rij);
					if (r2 < params.smoothingRadiusPow2)
					{
						const float r = sqrtf(r2);
						const float sRmin_r = params.smoothingRadius - r;
						const vec3 rijNorm = rij / r;
						const float pressureSum = pi.pressure + pj.pressure;
						const vec3 delta_v = pj.velocity - pi.velocity;
						const float fourtyFiveOverPI_SR6 = 45.0f / (glm::pi<float>() * params.smoothingRadiusPow6);

						forcePressure += -rijNorm * params.particleMass * pressureSum / (2.0f * pj.density) *
										 fourtyFiveOverPI_SR6 * sRmin_r * sRmin_r;
						forceViscosity += params.particleViscosity * params.particleMass * delta_v / pj.density *
										  fourtyFiveOverPI_SR6 * sRmin_r;
					}
				}
			}
		}
	}

	const vec3 forceGravity = -params.gravity * pi.density * params.gravityMult;
	pi.forcePhysic = forcePressure + forceViscosity + forceGravity;
}

__global__ void integrateAndCollisions(Particle *particles, SimulationParams params, Plane *planes, int particleCount,
									   int planeCount, float timestep, int *gridCounter, int *gridIndices,
									   vec3 worldMin, vec3 worldMax)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= particleCount)
		return;
	auto &pi = particles[idx];

	pi.velocity += timestep * pi.forcePhysic / pi.density;
	pi.position += timestep * pi.velocity;

	for (int i = 0; i < planeCount; i++)
	{
		vec3 penetrationNormal, penetrationPosition;
		float penetrationLength;
		if (plane_intersect(planes[i], pi.position, params.particleRadius, penetrationNormal, penetrationPosition,
							penetrationLength))
		{
			pi.velocity = dampVelocity(planes[i], pi.velocity, penetrationNormal, 1.0f - params.particleDrag);
			pi.position = penetrationPosition - penetrationNormal * fabs(penetrationLength);
		}
	}
}

// buildGrid();

void LaunchKernels(Particle *particles, int particleCount, SimulationParams params, Plane *planes, int planeCount,
				   float timestep, int *gridCounts, int *gridIndices, vec3 worldMin, vec3 worldMax)
{
	dim3 dimBlock = dim3(4, 4, 4);
	dim3 dimGrid = dim3(gridDimX + dimBlock.x - 1, gridDimY + dimBlock.y - 1, gridDimZ + dimBlock.z - 1);

	clearGrid<<<dimGrid, dimBlock>>>(gridCounts, gridIndices);

	dimBlock = dim3(16 * 16, 1, 1);
	dimGrid = dim3(particleCount + dimBlock.x - 1, 1, 1);
	buildGrid<<<dimGrid, dimBlock>>>(particles, particleCount, gridCounts, gridIndices, worldMin, worldMax);

	computeDensityPressure<<<dimGrid, dimBlock>>>(particles, params, planes, particleCount, planeCount, DT, gridCounts,
												  gridIndices, worldMin, worldMax);
	computeForces<<<dimGrid, dimBlock>>>(particles, params, planes, particleCount, planeCount, DT, gridCounts,
										 gridIndices, worldMin, worldMax);
	hipDeviceSynchronize();

	integrateAndCollisions<<<dimGrid, dimBlock>>>(particles, params, planes, particleCount, planeCount, DT, gridCounts,
												  gridIndices, worldMin, worldMax);

	hipDeviceSynchronize();
}